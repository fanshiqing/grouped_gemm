#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <torch/torch.h>
#include <hipcub/hipcub.hpp>
#ifdef ENABLE_BF16
#include <hip/hip_bf16.h>
#endif

#include "hip/hip_runtime.h"
#include ""

#include "ATen/cuda/HIPContext.h"

#include "sinkhorn.h"
#include "cutlass_kernels/th_utils.h"
#include "cutlass_kernels/moe_gemm/moe_gemm_kernels.h"
#include "cutlass_kernels/moe_gemm/moe_permute_kernels.h"
#include "cutlass_kernels/moe_gemm/moe_gemm_utils.h"

using torch::Tensor;

namespace groupedgemmformoe {

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Gemm Helper
//
/////////////////////////////////////////////////////////////////////////////////////////////////

// act type, weight type
template <typename T, typename WeightType>
Tensor run_group_gemm_helper(Tensor    input_activations,
                             Tensor    fc1_expert_weights,
                             Tensor    tokens_per_expert,
                             bool      transB)
{
    const int gemm_m = input_activations.size(0);
    int gemm_n;
    if (transB) gemm_n = fc1_expert_weights.size(1);
    else gemm_n = fc1_expert_weights.size(2);
    const int gemm_k = input_activations.size(1);
    const int num_experts = tokens_per_expert.size(0);

    if (gemm_k & 0x7 != 0)
    {
        throw std::runtime_error("gemm_k of grouped gemm with variable M must be a multiple of 8.");
    }

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    int *tokens_per_expert_ptr = get_ptr<int>(tokens_per_expert);

    T *input_act_ptr = get_ptr<T>(input_activations);
    WeightType *fc1_expert_weights_ptr = get_ptr<WeightType>(fc1_expert_weights);

    const at::ScalarType _st = input_activations.scalar_type();
    auto fc1_output =
        torch::empty({gemm_m, gemm_n}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    T *fc1_output_ptr = get_ptr<T>(fc1_output);

    groupedgemmformoe::MoeGemmRunner<T, WeightType> moe_gemm_runner_;

    moe_gemm_runner_.moe_gemm(input_act_ptr,
                              fc1_expert_weights_ptr,
                              fc1_output_ptr,
                              tokens_per_expert_ptr, // gemm_m
                              gemm_n,                // gemm_n
                              gemm_k,                // gemm_k
                              gemm_m,                // num_tokens
                              num_experts,
                              transB,
                              stream);

    return fc1_output;
}

// act type, weight type
template <typename T, typename WeightType>
Tensor run_group_gemm_backward_helper(Tensor input_activations,
                                      Tensor fc1_expert_weights,
                                      Tensor tokens_per_expert,
                                      bool   transC)
{
    // Matrix A: X      shape(m, k)
    // Matrix B: dL/dY  shape(m, n)
    // Output C: dL/dW  shape(k, n)

    const int gemm_m = input_activations.size(1);
    const int gemm_n = fc1_expert_weights.size(1);
    const int gemm_k = input_activations.size(0);
    const int num_experts = tokens_per_expert.size(0);

    if ((gemm_m & 0x7 != 0) || (gemm_n & 0x7 != 0))
    {
        throw std::runtime_error("gemm_m and gemm_n of grouped gemm with variable K must be multiples of 8.");
    }

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    int *tokens_per_expert_ptr = get_ptr<int>(tokens_per_expert);

    T *input_act_ptr = get_ptr<T>(input_activations);
    WeightType *fc1_expert_weights_ptr = get_ptr<WeightType>(fc1_expert_weights);

    const at::ScalarType _st = input_activations.scalar_type();
    Tensor fc1_output;
    if (transC)
    {
        fc1_output = torch::empty({num_experts, gemm_n, gemm_m}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    }
    else
    {
        fc1_output = torch::empty({num_experts, gemm_m, gemm_n}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    }
    T *fc1_output_ptr = get_ptr<T>(fc1_output);

    groupedgemmformoe::MoeGemmRunner<T, WeightType> moe_gemm_runner_;

    moe_gemm_runner_.moe_gemm_backward(input_act_ptr,
                                       fc1_expert_weights_ptr,
                                       fc1_output_ptr,
                                       gemm_m,                // gemm_m
                                       gemm_n,                // gemm_n
                                       tokens_per_expert_ptr, // gemm_k
                                       gemm_k,                // num_tokens
                                       num_experts,
                                       transC,
                                       stream);

    return fc1_output;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Grouped GEMM OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

Tensor moe_group_gemm_op(Tensor  input_activations,
                         Tensor  fc1_expert_weights,
                         Tensor  tokens_per_expert,
                         bool    transB)
{
    Tensor output_tensor;

    // activations type
    const at::ScalarType _st = input_activations.scalar_type();
    switch (_st) {
        case at::ScalarType::Float: {
            output_tensor = run_group_gemm_helper<float, float>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transB);
            break;
        }
        case at::ScalarType::Half: {
            output_tensor = run_group_gemm_helper<half, half>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transB);
            break;
        }
#ifdef ENABLE_BF16
        case at::ScalarType::BFloat16: {
            output_tensor = run_group_gemm_helper<__hip_bfloat16, __hip_bfloat16>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transB);
            break;
        }
#endif
        default:
            throw std::runtime_error("Wrong activation tensor type.");
    }
    return output_tensor;
}

Tensor moe_group_gemm_backward_op(Tensor input_activations,
                                  Tensor fc1_expert_weights,
                                  Tensor tokens_per_expert,
                                  bool   transC)
{
    Tensor output_tensor;

    // activations type
    const at::ScalarType _st = input_activations.scalar_type();
    switch (_st) {
        case at::ScalarType::Float: {
            output_tensor = run_group_gemm_backward_helper<float, float>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC);

            break;
        }
        case at::ScalarType::Half: {
            output_tensor = run_group_gemm_backward_helper<half, half>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC);

            break;
        }
#ifdef ENABLE_BF16
        case at::ScalarType::BFloat16: {
            output_tensor = run_group_gemm_backward_helper<__hip_bfloat16, __hip_bfloat16>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC);

            break;
        }
#endif
        default:
            throw std::runtime_error("Wrong activation tensor type.");
    }
    return output_tensor;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Permute OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

std::tuple<torch::Tensor, torch::Tensor, std::vector<Tensor>> moe_permute_op(
    Tensor original_input,
    Tensor expert_for_rows,
    std::vector<Tensor> workspace,
    int64_t max_token_num)
{
    // initialize the workspace on the first run
    if (workspace.empty()) {
        auto options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false);
        Tensor row_id = torch::range(0, max_token_num - 1, 1, options);
        Tensor sorted_expert_for_rows = torch::empty(max_token_num, options);
        Tensor dest_row_to_source_row = torch::empty(max_token_num, options);
        int *expert_for_rows_ptr = get_ptr<int>(expert_for_rows);
        int *row_id_ptr = get_ptr<int>(row_id);
        int *sorted_expert_for_rows_ptr = get_ptr<int>(sorted_expert_for_rows);
        int *dest_row_to_source_row_ptr = get_ptr<int>(dest_row_to_source_row);

        size_t temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes,
                                        expert_for_rows_ptr, sorted_expert_for_rows_ptr,
                                        row_id_ptr, dest_row_to_source_row_ptr, max_token_num);
        Tensor temp_storage = 
            torch::empty(temp_storage_bytes, torch::dtype(torch::kInt8).device(torch::kCUDA).requires_grad(false));

        workspace.push_back(row_id);
        workspace.push_back(sorted_expert_for_rows);
        workspace.push_back(dest_row_to_source_row);
        workspace.push_back(temp_storage);
    }

    int *expert_for_rows_ptr = get_ptr<int>(expert_for_rows);
    int *row_id_ptr = get_ptr<int>(workspace[0]);
    int *sorted_expert_for_rows_ptr = get_ptr<int>(workspace[1]);
    int *dest_row_to_source_row_ptr = get_ptr<int>(workspace[2]);

    const int num_rows = original_input.size(0);
    const int num_cols = original_input.size(1);

    // Run sorting operation
    void *d_temp_storage = get_ptr<void>(workspace[3]);
    size_t temp_storage_bytes = std::numeric_limits<size_t>::max();
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    expert_for_rows_ptr, sorted_expert_for_rows_ptr,
                                    row_id_ptr, dest_row_to_source_row_ptr, num_rows);

    // activations type
    const at::ScalarType _st = original_input.scalar_type();

    // Output buffer alloc
    Tensor permuted_output =
        torch::empty({num_rows, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    Tensor source_row_to_dest_row = 
        torch::empty(num_rows, torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

    int *&map_dest_row_to_source_row = dest_row_to_source_row_ptr;
    int *map_source_row_to_dest_row = get_ptr<int>(source_row_to_dest_row);;

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType>(
            original_input_ptr,
            permuted_output_ptr,
            map_dest_row_to_source_row,
            map_source_row_to_dest_row,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = half;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType>(
            original_input_ptr,
            permuted_output_ptr,
            map_dest_row_to_source_row,
            map_source_row_to_dest_row,
            num_rows,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = __hip_bfloat16;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType>(
            original_input_ptr,
            permuted_output_ptr,
            map_dest_row_to_source_row,
            map_source_row_to_dest_row,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    /// Removed to align with pytorch
    // hipStreamSynchronize(stream);

    return std::make_tuple(permuted_output, source_row_to_dest_row, workspace);
}

torch::Tensor moe_recover_op(
    Tensor permuted_input,
    Tensor source_row_to_dest_row)
{
    const int num_rows = permuted_input.size(0);
    const int num_cols = permuted_input.size(1);

    // activations type
    const at::ScalarType _st = permuted_input.scalar_type();

    // Output buffer alloc
    Tensor unpermuted_output =
        torch::empty({num_rows, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));

    int *map_source_row_to_dest_row = get_ptr<int>(source_row_to_dest_row);
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            map_source_row_to_dest_row,
            nullptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = half;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            map_source_row_to_dest_row,
            nullptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = __hip_bfloat16;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            map_source_row_to_dest_row,
            nullptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    /// Removed to align with pytorch
    // hipStreamSynchronize(stream);

    return unpermuted_output;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// TORCH_LIBRARY
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TORCH_LIBRARY(moe_unit_ops, m)
{
    m.def("moe_group_gemm_op", moe_group_gemm_op);
    m.def("moe_group_gemm_backward_op", moe_group_gemm_backward_op);
    m.def("moe_permute_op", moe_permute_op);
    m.def("moe_recover_op", moe_recover_op);
    // TODO: find a more reasonable repo to place this kernel.
    m.def("sinkhorn", sinkhorn);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
} // namespace groupedgemmformoe