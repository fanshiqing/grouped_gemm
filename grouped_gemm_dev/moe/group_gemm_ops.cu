#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <torch/torch.h>
#include <hipcub/hipcub.hpp>
#ifdef ENABLE_BF16
#include <hip/hip_bf16.h>
#endif

#include "hip/hip_runtime.h"
#include ""

#include "ATen/cuda/HIPContext.h"

#include "sinkhorn.h"
#include "cutlass_kernels/th_utils.h"
#include "cutlass_kernels/moe_gemm/moe_gemm_kernels.h"
#include "cutlass_kernels/moe_gemm/moe_gemm_backward_template.h"
#include "cutlass_kernels/moe_gemm/moe_permute_kernels.h"
#include "cutlass_kernels/moe_gemm/moe_gemm_utils.h"

#include "cublas_wrapper.h"

using torch::Tensor;

bool USE_CUBLAS = false;

namespace groupedgemmformoe {

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Gemm Helper
//
/////////////////////////////////////////////////////////////////////////////////////////////////

// act type, weight type
template <typename T, typename WeightType>
Tensor run_group_gemm_helper(Tensor              input_activations,
                             std::vector<Tensor> fc1_expert_weights_list,
                             Tensor              tokens_per_expert,
                             bool                transB)
{
    const int gemm_m = input_activations.size(0);
    int gemm_n;
    if (transB) gemm_n = fc1_expert_weights_list[0].size(0);
    else gemm_n = fc1_expert_weights_list[0].size(1);
    const int gemm_k = input_activations.size(1);
    const int num_experts = tokens_per_expert.size(0);

    if (gemm_k & 0x7 != 0)
    {
        throw std::runtime_error("gemm_k of grouped gemm with variable M must be a multiple of 8.");
    }

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    int *tokens_per_expert_ptr = get_ptr<int>(tokens_per_expert);

    T *input_act_ptr = get_ptr<T>(input_activations);
    WeightType *fc1_expert_weights_ptr_list[num_experts];
    for (size_t i = 0; i < num_experts; i++)
    {
        fc1_expert_weights_ptr_list[i] = get_ptr<WeightType>(fc1_expert_weights_list[i]);
    }

    const at::ScalarType _st = input_activations.scalar_type();
    auto fc1_output =
        torch::empty({gemm_m, gemm_n}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    T *fc1_output_ptr = get_ptr<T>(fc1_output);

    int sm_ = getSMVersion();

    if (sm_ != 90 && (USE_CUBLAS == false))
    {
        groupedgemmformoe::MoeGemmRunner<T, WeightType> moe_gemm_runner_;

        moe_gemm_runner_.moe_gemm(input_act_ptr,
                                  fc1_expert_weights_ptr_list,
                                  fc1_output_ptr,
                                  tokens_per_expert_ptr, // gemm_m
                                  gemm_n,                // gemm_n
                                  gemm_k,                // gemm_k
                                  gemm_m,                // num_tokens
                                  num_experts,
                                  transB,
                                  stream);
    }
    else
    {
        if (!cublas_init)
            cublas_handle_init();

        cublas_group_gemm_helper<T>(
            input_act_ptr,
            fc1_expert_weights_ptr_list,
            fc1_output_ptr,
            tokens_per_expert_ptr, // gemm_m
            gemm_n,                // gemm_n
            gemm_k,                // gemm_k
            num_experts,
            transB,
            stream);
    }

    return fc1_output;
}

template <typename T,
          typename WeightType,
          typename AccumGradType>
void group_gemm_varK_algo_dispatcher(T*              A,
                                     WeightType*     B,
                                     T*              C,
                                     AccumGradType** weight_grad_list,
                                     int64_t         gemm_m,
                                     int64_t         gemm_n,
                                     int*            gemm_k_per_expert,
                                     int             num_tokens,
                                     int             num_experts,
                                     bool            transC,
                                     hipStream_t    stream)
{
    int sm_ = getSMVersion();

    if (sm_ != 90)
    {
        groupedgemmformoe::MoeGemmRunner<T, WeightType> moe_gemm_runner_;

        moe_gemm_runner_.template moe_gemm_backward<AccumGradType>(
            A,
            B,
            C,
            weight_grad_list,
            gemm_m,
            gemm_n,
            gemm_k_per_expert,
            num_tokens,
            num_experts,
            transC,
            stream);
    }
    else
    {
        if (!cublas_init)
            cublas_handle_init();

        cublas_group_gemm_helper<T, AccumGradType>(
            A,
            B,
            C,
            weight_grad_list,
            gemm_m,
            gemm_n,
            gemm_k_per_expert,
            num_experts,
            transC,
            stream);
    }
}

// act type, weight type
template <typename T, typename WeightType>
Tensor run_group_gemm_backward_helper(Tensor input_activations,
                                      Tensor fc1_expert_weights,
                                      Tensor tokens_per_expert,
                                      bool   transC,
                                      std::vector<Tensor> weight_grad_list)
{
    // Matrix A: X      shape(m, k)
    // Matrix B: dL/dY  shape(m, n)
    // Output C: dL/dW  shape(k, n)

    const int gemm_m = input_activations.size(1);
    const int gemm_n = fc1_expert_weights.size(1);
    const int gemm_k = input_activations.size(0);
    const int num_experts = tokens_per_expert.size(0);

    if ((gemm_m & 0x7 != 0) || (gemm_n & 0x7 != 0))
    {
        throw std::runtime_error("gemm_m and gemm_n of grouped gemm with variable K must be multiples of 8.");
    }

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    int *tokens_per_expert_ptr = get_ptr<int>(tokens_per_expert);

    T *input_act_ptr = get_ptr<T>(input_activations);
    WeightType *fc1_expert_weights_ptr = get_ptr<WeightType>(fc1_expert_weights);

    const at::ScalarType _st = input_activations.scalar_type();
    Tensor fc1_output;

    if (weight_grad_list.empty())
    {
        if (transC)
        {
            fc1_output = torch::empty({num_experts, gemm_n, gemm_m}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
        }
        else
        {
            fc1_output = torch::empty({num_experts, gemm_m, gemm_n}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
        }

        T *fc1_output_ptr = get_ptr<T>(fc1_output);
        group_gemm_varK_algo_dispatcher<T, WeightType, T>(
            input_act_ptr,
            fc1_expert_weights_ptr,
            fc1_output_ptr,
            nullptr,
            gemm_m,                // gemm_m
            gemm_n,                // gemm_n
            tokens_per_expert_ptr, // gemm_k
            gemm_k,                // num_tokens
            num_experts,
            transC,
            stream);
    }
    else
    {
        const at::ScalarType _st = weight_grad_list[0].scalar_type();
        switch (_st) {
            case at::ScalarType::Float: {
                using dType = float;

                dType *weight_grad_ptr_list[num_experts];
                for (size_t i = 0; i < num_experts; i++)
                {
                    weight_grad_ptr_list[i] = get_ptr<dType>(weight_grad_list[i]);
                }

                group_gemm_varK_algo_dispatcher<T, WeightType, dType>(
                    input_act_ptr,
                    fc1_expert_weights_ptr,
                    nullptr,
                    weight_grad_ptr_list,
                    gemm_m,                // gemm_m
                    gemm_n,                // gemm_n
                    tokens_per_expert_ptr, // gemm_k
                    gemm_k,                // num_tokens
                    num_experts,
                    transC,
                    stream);

                break;
            }
            case at::ScalarType::Half: {
                using dType = half;

                dType *weight_grad_ptr_list[num_experts];
                for (size_t i = 0; i < num_experts; i++)
                {
                    weight_grad_ptr_list[i] = get_ptr<dType>(weight_grad_list[i]);
                }

                group_gemm_varK_algo_dispatcher<T, WeightType, dType>(
                    input_act_ptr,
                    fc1_expert_weights_ptr,
                    nullptr,
                    weight_grad_ptr_list,
                    gemm_m,                // gemm_m
                    gemm_n,                // gemm_n
                    tokens_per_expert_ptr, // gemm_k
                    gemm_k,                // num_tokens
                    num_experts,
                    transC,
                    stream);

                break;
            }
#ifdef ENABLE_BF16
            case at::ScalarType::BFloat16: {
                using dType = __hip_bfloat16;

                dType *weight_grad_ptr_list[num_experts];
                for (size_t i = 0; i < num_experts; i++)
                {
                    weight_grad_ptr_list[i] = get_ptr<dType>(weight_grad_list[i]);
                }

                group_gemm_varK_algo_dispatcher<T, WeightType, dType>(
                    input_act_ptr,
                    fc1_expert_weights_ptr,
                    nullptr,
                    weight_grad_ptr_list,
                    gemm_m,                // gemm_m
                    gemm_n,                // gemm_n
                    tokens_per_expert_ptr, // gemm_k
                    gemm_k,                // num_tokens
                    num_experts,
                    transC,
                    stream);

                break;
            }
#endif
            default:
                throw std::runtime_error("Wrong main_grad tensor data type.");
        }
    }

    return fc1_output;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Grouped GEMM OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

Tensor moe_group_gemm_op(Tensor              input_activations,
                         std::vector<Tensor> fc1_expert_weights_list,
                         Tensor              tokens_per_expert,
                         bool                transB)
{
    Tensor output_tensor;

    // activations type
    const at::ScalarType _st = input_activations.scalar_type();
    switch (_st) {
        case at::ScalarType::Float: {
            output_tensor = run_group_gemm_helper<float, float>(
                input_activations,
                fc1_expert_weights_list,
                tokens_per_expert,
                transB);
            break;
        }
        case at::ScalarType::Half: {
            output_tensor = run_group_gemm_helper<half, half>(
                input_activations,
                fc1_expert_weights_list,
                tokens_per_expert,
                transB);
            break;
        }
#ifdef ENABLE_BF16
        case at::ScalarType::BFloat16: {
            output_tensor = run_group_gemm_helper<__hip_bfloat16, __hip_bfloat16>(
                input_activations,
                fc1_expert_weights_list,
                tokens_per_expert,
                transB);
            break;
        }
#endif
        default:
            throw std::runtime_error("Wrong activation tensor type.");
    }
    return output_tensor;
}

Tensor moe_group_gemm_backward_op(Tensor input_activations,
                                  Tensor fc1_expert_weights,
                                  Tensor tokens_per_expert,
                                  bool   transC,
                                  std::vector<Tensor> weight_grad_list)
{
    Tensor output_tensor;

    // activations type
    const at::ScalarType _st = input_activations.scalar_type();
    switch (_st) {
        case at::ScalarType::Float: {
            output_tensor = run_group_gemm_backward_helper<float, float>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC,
                weight_grad_list);

            break;
        }
        case at::ScalarType::Half: {
            output_tensor = run_group_gemm_backward_helper<half, half>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC,
                weight_grad_list);

            break;
        }
#ifdef ENABLE_BF16
        case at::ScalarType::BFloat16: {
            output_tensor = run_group_gemm_backward_helper<__hip_bfloat16, __hip_bfloat16>(
                input_activations,
                fc1_expert_weights,
                tokens_per_expert,
                transC,
                weight_grad_list);

            break;
        }
#endif
        default:
            throw std::runtime_error("Wrong activation tensor type.");
    }
    return output_tensor;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Permute OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

std::tuple<torch::Tensor, torch::Tensor, std::vector<Tensor>> moe_permute_op(
    Tensor original_input,
    Tensor expert_for_rows,
    Tensor row_id_map,
    std::vector<Tensor> workspace,
    int64_t max_token_num)
{
    const int num_rows = original_input.size(0);
    const int num_cols = original_input.size(1);

    // activations type
    const at::ScalarType _st = original_input.scalar_type();

    if (!row_id_map.defined()) {
        // initialize the workspace on the first run
        if (workspace.empty()) {
            auto options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false);
            Tensor row_id = torch::range(0, max_token_num - 1, 1, options);
            Tensor sorted_expert_for_rows = torch::empty(max_token_num, options);

            size_t temp_storage_bytes = 0;
            int *temp_ptr = nullptr;
            hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes,
                                            temp_ptr, temp_ptr,
                                            temp_ptr, temp_ptr, max_token_num);
            Tensor temp_storage = 
                torch::empty(temp_storage_bytes, torch::dtype(torch::kInt8).device(torch::kCUDA).requires_grad(false));

            workspace.push_back(row_id);
            workspace.push_back(sorted_expert_for_rows);
            workspace.push_back(temp_storage);
        }

        // Output buffer alloc
        row_id_map = 
            torch::empty(num_rows, torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

        int *expert_for_rows_ptr = get_ptr<int>(expert_for_rows);
        int *row_id_ptr = get_ptr<int>(workspace[0]);
        int *sorted_expert_for_rows_ptr = get_ptr<int>(workspace[1]);
        int *row_id_map_ptr = get_ptr<int>(row_id_map);

        // Run sorting operation
        void *d_temp_storage = get_ptr<void>(workspace[2]);
        size_t temp_storage_bytes = std::numeric_limits<size_t>::max();
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                        expert_for_rows_ptr, sorted_expert_for_rows_ptr,
                                        row_id_ptr, row_id_map_ptr, num_rows);
    }

    // Output buffer alloc
    Tensor permuted_output =
        torch::empty({num_rows, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));

    int *row_id_map_ptr = get_ptr<int>(row_id_map);
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType, true, 4>(
            original_input_ptr,
            permuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = half;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType, true, 8>(
            original_input_ptr,
            permuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = __hip_bfloat16;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType, true, 8>(
            original_input_ptr,
            permuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
#ifdef ENABLE_FP8
    case at::ScalarType::Float8_e5m2:
    {
        using dType = __hip_fp8_e5m2_fnuz;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType, true, 16>(
            original_input_ptr,
            permuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Float8_e4m3fn:
    {
        using dType = __hip_fp8_e4m3_fnuz;

        dType *original_input_ptr = get_ptr<dType>(original_input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_kernel_launcher<dType, true, 16>(
            original_input_ptr,
            permuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    /// Removed to align with pytorch
    // hipStreamSynchronize(stream);

    return std::make_tuple(permuted_output, row_id_map, workspace);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Unpermute OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

torch::Tensor moe_recover_op(
    Tensor permuted_input,
    Tensor row_id_map)
{
    const int num_rows = permuted_input.size(0);
    const int num_cols = permuted_input.size(1);

    // activations type
    const at::ScalarType _st = permuted_input.scalar_type();

    // Output buffer alloc
    Tensor unpermuted_output =
        torch::empty({num_rows, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));

    int *row_id_map_ptr = get_ptr<int>(row_id_map);
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType, false, 4>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = half;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType, false, 8>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = __hip_bfloat16;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType, false, 8>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
#ifdef ENABLE_FP8
    case at::ScalarType::Float8_e5m2:
    {
        using dType = __hip_fp8_e5m2_fnuz;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType, false, 16>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Float8_e4m3fn:
    {
        using dType = __hip_fp8_e4m3_fnuz;

        dType *permuted_input_ptr = get_ptr<dType>(permuted_input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_kernel_launcher<dType, false, 16>(
            permuted_input_ptr,
            unpermuted_output_ptr,
            row_id_map_ptr,
            num_rows,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    /// Removed to align with pytorch
    // hipStreamSynchronize(stream);

    return unpermuted_output;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Permute_topK OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

std::tuple<torch::Tensor, torch::Tensor, std::vector<Tensor>> moe_permute_topK_op(
    Tensor              input,
    Tensor              indices,
    std::vector<Tensor> workspace,
    int64_t             max_expanded_token_num)
{
    const int num_tokens = input.size(0);
    const int num_cols = input.size(1);
    const int num_topK = indices.size(1);

    // initialize the workspace on the first run
    if (workspace.empty()) {
        auto options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false);

        Tensor sorted_indices = torch::empty(max_expanded_token_num, options);
        Tensor row_id = torch::range(0, max_expanded_token_num - 1, 1, options);
        Tensor sorted_row_id =
            torch::empty(max_expanded_token_num, torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

        size_t temp_storage_bytes = 0;
        int *temp_ptr = nullptr;
        hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes,
                                        temp_ptr, temp_ptr,
                                        temp_ptr, temp_ptr, max_expanded_token_num);
        Tensor temp_storage =
            torch::empty(temp_storage_bytes, torch::dtype(torch::kInt8).device(torch::kCUDA).requires_grad(false));

        workspace.push_back(sorted_indices);
        workspace.push_back(row_id);
        workspace.push_back(sorted_row_id);
        workspace.push_back(temp_storage);
    }

    int *indices_ptr = get_ptr<int>(indices);
    int *sorted_indices_ptr = get_ptr<int>(workspace[0]);
    int *row_id_ptr = get_ptr<int>(workspace[1]);
    int *sorted_row_id_ptr = get_ptr<int>(workspace[2]);

    void *d_temp_storage = get_ptr<void>(workspace[3]);
    size_t temp_storage_bytes = std::numeric_limits<size_t>::max();

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    indices_ptr, sorted_indices_ptr,
                                    row_id_ptr, sorted_row_id_ptr, num_tokens * num_topK);

    // activations type
    const at::ScalarType _st = input.scalar_type();

    // Output buffer alloc
    Tensor permuted_output =
        torch::empty({num_tokens * num_topK, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    Tensor row_id_map = 
        torch::empty({num_tokens * num_topK}, torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

    int *row_id_map_ptr = get_ptr<int>(row_id_map);
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;
        using dTypeCompute = float;

        dType *input_ptr = get_ptr<dType>(input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 4>(
            input_ptr,
            permuted_output_ptr,
            sorted_row_id_ptr,
            row_id_map_ptr,
            nullptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = cutlass::half_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 8>(
            input_ptr,
            permuted_output_ptr,
            sorted_row_id_ptr,
            row_id_map_ptr,
            nullptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = cutlass::bfloat16_t;
        using dTypeCompute = cutlass::bfloat16_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 8>(
            input_ptr,
            permuted_output_ptr,
            sorted_row_id_ptr,
            row_id_map_ptr,
            nullptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#endif
#ifdef ENABLE_FP8
    case at::ScalarType::Float8_e5m2:
    {
        using dType = cutlass::float_e5m2_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 16>(
            input_ptr,
            permuted_output_ptr,
            sorted_row_id_ptr,
            row_id_map_ptr,
            nullptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Float8_e4m3fn:
    {
        using dType = cutlass::float_e4m3_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *permuted_output_ptr = get_ptr<dType>(permuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 16>(
            input_ptr,
            permuted_output_ptr,
            sorted_row_id_ptr,
            row_id_map_ptr,
            nullptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    return std::make_tuple(permuted_output, row_id_map, workspace);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Unpermute_topK OP
//
/////////////////////////////////////////////////////////////////////////////////////////////////

torch::Tensor moe_recover_topK_op(
    Tensor  input,
    Tensor  row_id_map,
    Tensor  prob,
    int64_t num_tokens,
    int64_t num_topK)
{
    const int num_cols = input.size(1);

    // activations type
    const at::ScalarType _st = input.scalar_type();

    // Output buffer alloc
    Tensor unpermuted_output =
        torch::empty({num_tokens, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));

    int *row_id_map_ptr = get_ptr<int>(row_id_map);
    float *prob_ptr = (prob.defined()) ? get_ptr<float>(prob) : nullptr;
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;
        using dTypeCompute = float;

        dType *input_ptr = get_ptr<dType>(input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, false, 4>(
            input_ptr,
            unpermuted_output_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = cutlass::half_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, false, 8>(
            input_ptr,
            unpermuted_output_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = cutlass::bfloat16_t;
        using dTypeCompute = cutlass::bfloat16_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, false, 8>(
            input_ptr,
            unpermuted_output_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#endif
#ifdef ENABLE_FP8
    case at::ScalarType::Float8_e5m2:
    {
        using dType = cutlass::float_e5m2_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, false, 16>(
            input_ptr,
            unpermuted_output_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
    case at::ScalarType::Float8_e4m3fn:
    {
        using dType = cutlass::float_e4m3_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_ptr = get_ptr<dType>(input);
        dType *unpermuted_output_ptr = get_ptr<dType>(unpermuted_output);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, false, 16>(
            input_ptr,
            unpermuted_output_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    return unpermuted_output;
}

std::tuple<torch::Tensor, torch::Tensor> moe_recover_topK_bwd_op(
    Tensor  input_bwd,
    Tensor  input_fwd,
    Tensor  row_id_map,
    Tensor  prob)
{
    const int num_tokens = prob.size(0);
    const int num_topK = prob.size(1);
    const int num_cols = input_bwd.size(1);

    int *row_id_map_ptr = get_ptr<int>(row_id_map);
    float *prob_ptr = get_ptr<float>(prob);

    // activations type
    const at::ScalarType _st = input_bwd.scalar_type();

    // Output buffer alloc
    Tensor act_grad =
        torch::empty({num_tokens * num_topK, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
    Tensor prob_grad =
        torch::empty({num_tokens, num_topK}, torch::dtype(torch::kFloat32).device(torch::kCUDA).requires_grad(false));
    float *prob_grad_ptr = get_ptr<float>(prob_grad);

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    switch (_st)
    {
    case at::ScalarType::Float:
    {
        using dType = float;
        using dTypeCompute = float;

        dType *input_bwd_ptr = get_ptr<dType>(input_bwd);
        dType *input_fwd_ptr = get_ptr<dType>(input_fwd);
        dType *act_grad_ptr = get_ptr<dType>(act_grad);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 4>(
            input_bwd_ptr,
            act_grad_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream,
            prob_grad_ptr,
            input_fwd_ptr);

        break;
    }
    case at::ScalarType::Half:
    {
        using dType = cutlass::half_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_bwd_ptr = get_ptr<dType>(input_bwd);
        dType *input_fwd_ptr = get_ptr<dType>(input_fwd);
        dType *act_grad_ptr = get_ptr<dType>(act_grad);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 8>(
            input_bwd_ptr,
            act_grad_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream,
            prob_grad_ptr,
            input_fwd_ptr);

        break;
    }
#ifdef ENABLE_BF16
    case at::ScalarType::BFloat16:
    {
        using dType = cutlass::bfloat16_t;
        using dTypeCompute = cutlass::bfloat16_t;

        dType *input_bwd_ptr = get_ptr<dType>(input_bwd);
        dType *input_fwd_ptr = get_ptr<dType>(input_fwd);
        dType *act_grad_ptr = get_ptr<dType>(act_grad);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 8>(
            input_bwd_ptr,
            act_grad_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream,
            prob_grad_ptr,
            input_fwd_ptr);

        break;
    }
#endif
#ifdef ENABLE_FP8
    case at::ScalarType::Float8_e5m2:
    {
        using dType = cutlass::float_e5m2_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_bwd_ptr = get_ptr<dType>(input_bwd);
        dType *input_fwd_ptr = get_ptr<dType>(input_fwd);
        dType *act_grad_ptr = get_ptr<dType>(act_grad);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 16>(
            input_bwd_ptr,
            act_grad_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream,
            prob_grad_ptr,
            input_fwd_ptr);

        break;
    }
    case at::ScalarType::Float8_e4m3fn:
    {
        using dType = cutlass::float_e4m3_t;
        using dTypeCompute = cutlass::half_t;

        dType *input_bwd_ptr = get_ptr<dType>(input_bwd);
        dType *input_fwd_ptr = get_ptr<dType>(input_fwd);
        dType *act_grad_ptr = get_ptr<dType>(act_grad);

        moe_permute_topK_kernel_launcher<dType, dTypeCompute, true, 16>(
            input_bwd_ptr,
            act_grad_ptr,
            nullptr,
            row_id_map_ptr,
            prob_ptr,
            num_tokens,
            num_topK,
            num_cols,
            stream,
            prob_grad_ptr,
            input_fwd_ptr);

        break;
    }
#endif
    default:
        throw std::runtime_error("Wrong activation tensor type.");
    }

    return std::make_tuple(act_grad, prob_grad);
}

void use_cublas_for_groupedgemm(bool enable)
{
    USE_CUBLAS = enable;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// TORCH_LIBRARY
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TORCH_LIBRARY(moe_unit_ops, m)
{
    m.def("moe_group_gemm_op", moe_group_gemm_op);
    m.def("moe_group_gemm_backward_op", moe_group_gemm_backward_op);
    m.def("moe_permute_op", moe_permute_op);
    m.def("moe_recover_op", moe_recover_op);
    m.def("moe_permute_topK_op", moe_permute_topK_op);
    m.def("moe_recover_topK_op", moe_recover_topK_op);
    m.def("moe_recover_topK_bwd_op", moe_recover_topK_bwd_op);
    // TODO: find a more reasonable repo to place this kernel.
    m.def("sinkhorn", sinkhorn);
    m.def("use_cublas_for_groupedgemm", use_cublas_for_groupedgemm);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
} // namespace groupedgemmformoe